#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
using namespace std;
using namespace cv;
 
//Sobel operator edge detection kernel function
__global__ void sobelInCuda(unsigned char *dataIn, unsigned char *dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    int Gx = 0;
    int Gy = 0;
 
    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth + xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) * imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) * imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        dataOut[index] = (abs(Gx) + abs(Gy)) / 2;
    }
}
 
//Sobel operator edge detection CPU function
void sobel(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    int Gx = 0;
    int Gy = 0;
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar *dataUp = srcImg.ptr<uchar>(i - 1);
        uchar *data = srcImg.ptr<uchar>(i);
        uchar *dataDown = srcImg.ptr<uchar>(i + 1);
        uchar *out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            out[j] = (abs(Gx) + abs(Gy)) / 2;
        }
    }
}
 
int main()
{
    Mat grayImg = imread("1.jpg", 0);
 
    int imgHeight = grayImg.rows;
    int imgWidth = grayImg.cols;
 
    Mat gaussImg;
    //Gaussian filtering
    GaussianBlur(grayImg, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);
 
    //Sobel operator CPU implementation
    Mat dst(imgHeight, imgWidth, CV_8UC1, Scalar(0));
    sobel(gaussImg, dst, imgHeight, imgWidth);
 
    //The returned image after CUDA is implemented
    Mat dstImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));
 
    //Create GPU memory
    unsigned char *d_in;
    unsigned char *d_out;
 
    hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));
 
    //Pass the Gaussian filtered image from the CPU to the GPU
    hipMemcpy(d_in, gaussImg.data, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);
 
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);
 
    //Call kernel function
    sobelInCuda << <blocksPerGrid, threadsPerBlock >> >(d_in, d_out, imgHeight, imgWidth);
 
    //Pass the image back to the GPU
    hipMemcpy(dstImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);
 
    //Free GPU memory
    hipFree(d_in);
    hipFree(d_out);
 
    return 0;
}
